// %RANGE% TUNE_ITEMS ipt 7:24:1
// %RANGE% TUNE_THREADS tpb 128:1024:32
// %RANGE% CUB_DETAIL_L2_BACKOFF_NS l2b 0:1200:5
// %RANGE% CUB_DETAIL_L2_WRITE_LATENCY_NS l2w 0:1200:5

#include <nvbench_helper.cuh>

using op_t = hipcub::Sum;
#include "base.cuh"
